#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define GPU_RUNS 300
#define ELEMENTS_PER_THREAD 10
#define NUM_BITS 4
#define H (1 << NUM_BITS)

#include "host_skel.cuh"
#include "helper.h"
#include "kernels.cuh"

// *Very* beautiful binary printer:
void binaryPrinter(int val, unsigned int decimal_points) {
    for (int i = decimal_points-1; i >= 0; i--) {
        if (val & (1 << i)) {
            printf("1");
        }
        else {
            printf("0");
        }
    }
}

// Modified From assignment 2:
void scanIncAddI32(const uint32_t B     // desired CUDA block size ( <= 1024, multiple of 32)
                 , const size_t   N     // length of the input array
                 , uint32_t* d_in            // device input  of size: N * sizeof(uint32_t)
                 , uint32_t* d_out           // device result of size: N * sizeof(uint32_t)
) {
    uint32_t* d_tmp;
    hipMalloc((void**)&d_tmp, MAX_BLOCK*sizeof(uint32_t));
    hipMemset(d_out, 0, N*sizeof(uint32_t));

    scanInc<Add<uint32_t>> ( B, N, d_out, d_in, d_tmp );

    hipFree(d_tmp);
}

// Modified from assignment 3-4:
/**
 * Input:
 *   inp_d : [height][width]uint32_t
 * Result:
 *   out_d : [width][height]uint32_t
 *   (the transpose of inp_d.)
 */
template<int T>
void callTransposeKer( uint32_t*          inp_d,  
                       uint32_t*          out_d, 
                       const uint32_t height, 
                       const uint32_t width
) {
    // 1. setup block and grid parameters
    int  dimy = (height+T-1) / T; 
    int  dimx = (width +T-1) / T;
    dim3 block(T, T, 1);
    dim3 grid (dimx, dimy, 1);

    //2. execute the kernel
    coalsTransposeKer<T> <<< grid, block >>>(inp_d, out_d, height, width);
}

int main(int argc, char** argv) {
    uint32_t N;

    initHwd();
    
    { // reading the number of elements 
        if (argc != 2) { 
            printf("Num Args is: %d instead of 1. Exiting!\n", argc); 
            exit(1);
        }
        
        N = (uint32_t)atoi(argv[1]);
        printf("N is: %d\n", N);
        
        const uint32_t maxN = 500000000;
        if(N > maxN) {
            printf("N is too big; maximal value is %d. Exiting!\n", maxN);
            exit(2);
        }
    }
    
    // use the first CUDA device:
    hipSetDevice(0);
    
    uint32_t Q = 1;
    unsigned int B = 16;
    unsigned int numblocks = (N + (Q * B - 1)) / (Q * B);
    printf("Num blocks: %d \n", numblocks);
    unsigned int mask = (1 << NUM_BITS) - 1; // 4 bits = 0xF for radix 16

    uint32_t mem_size = N * sizeof(uint32_t);
    uint32_t hist_size = numblocks * H * sizeof(uint32_t);
    printf("Mem size: %d: ", mem_size);
    printf("Hist size: %d: ", hist_size);

    // allocate host memory for both CPU and GPU
    uint32_t* h_in  = (uint32_t*) malloc(mem_size);
    uint32_t* gpu_res = (uint32_t*) malloc(hist_size);
    
    
    // initialize the memory
    srand(time(NULL));
    printf("Input: (N):\n");
    for(unsigned int i=0; i<N; ++i) {
        h_in[i] = (uint32_t)rand() % N; // values between 0 and N 
        binaryPrinter(h_in[i], NUM_BITS);
        printf(", ");
    }

    // allocate device memory
    uint32_t* d_in;
    uint32_t* d_hist;
    uint32_t* d_hist_buffer;
    hipMalloc((void**)&d_in,  mem_size);
    hipMalloc((void**)&d_hist, hist_size);
    hipMalloc((void**)&d_hist_buffer, hist_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);
    hipMemset(d_hist, 0, hist_size);
    
    // a small number of dry runs
    for(int r = 0; r < 1; r++) {
        dim3 block(B, 1, 1), grid(numblocks, 1, 1);
        histogramKer<<< grid, block>>>(d_in, d_hist, mask, Q, N);
    }

    {


        //The cpu does the following:
        //Holds the outer loop over passes (for pass in [0..num_passes))
        // Calculates mask and shift for each bit group
        // Launches the three GPU kernels per pass (histogram → scan → scatter) 
        // Swaps input/output pointers between passes

        //Allocates global memory buffers on device:
        // d_in, d_out for the arrays being sorted
        // d_histograms (size = numBlocks × H)
        // d_prefixes (prefix sums of histograms)
        // Performs small global memory resets (e.g. hipMemset)
        // Does NOT touch shared or register memory (that’s only inside kernels)

        // for(int r = 0; r < 1; r++) {
        histogramKer<<<numblocks, B>>>(d_in, d_hist, mask, Q, N);
        hipDeviceSynchronize();
        callTransposeKer<32>(d_hist, d_hist_buffer, numblocks, H); //Maybe use other B value here
        hipDeviceSynchronize();
        scanIncAddI32(B, numblocks * H, d_hist_buffer, d_hist);
        hipDeviceSynchronize();
        callTransposeKer<32>(d_hist, d_hist_buffer, H, numblocks);
        hipDeviceSynchronize();
        mask = mask << NUM_BITS;
        // }
    }
        
    // check for errors
    gpuAssert( hipPeekAtLastError() );

    // copy result from device to host
    hipMemcpy(gpu_res, d_hist_buffer, hist_size, hipMemcpyDeviceToHost);
    
    // element-wise compare of CPU and GPU execution
   for (int b = 0; b < numblocks; b++) {
    printf("\nBlock %d histogram:\n", b);
    for (int i = 0; i < H; i++)
        printf("%u ", gpu_res[b * H + i]);
    }

    printf("\nReached the end! ^_^ \n");

    // clean-up memory
    free(h_in);       free(gpu_res); 
    hipFree(d_in);   hipFree(d_hist);
    hipFree(d_hist_buffer);
}



/**
 Pizza:             Pepsi:`
 ---------------------------------------
      __________           
    // ^   .  O \\           ___;_
   ||..   O      ||         /_____\ 
   || O    . ^   ||         |     |
   ||   ^    . O ||         |Pepsi|
   ||.   ^  O    ||         |_____|
    \\__________//          \_____/
 ````````````````````````````````````````
 */