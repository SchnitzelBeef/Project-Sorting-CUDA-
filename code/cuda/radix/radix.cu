#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define GPU_RUNS 500
#define VERBOSE false

#include "host_skel.cuh"
#include "helper.h"
#include "kernels.cuh"

void handleArgs(int argc, char** argv, uint32_t& N, uint32_t& Q, uint32_t& B, uint32_t& NUM_BITS, bool& useFile);

void cubRadixSort(uint32_t* d_in, uint32_t* d_out, size_t N, timeval& t_start, timeval& t_end);

void scanIncAddI32(const uint32_t B, const size_t N, uint32_t* d_in, uint32_t* d_out);

template<int T>
void callTransposeKer(uint32_t* inp_d, uint32_t* out_d, const uint32_t height, const uint32_t width);

void binaryPrinter(int val, unsigned int decimal_points);

int main(int argc, char** argv) {
    // Arg1: N - number of elements (Required)
    // Arg2: Q - number of elements per thread (Optional) Default: 5
    // Arg3: B - number of threads per block (Optional) Default: 32
    // Arg4: b - number of bits per pass (Optional) Default: 2
    // Arg5: flag - use external input file (Optional) Default: false
    uint32_t N;
    // Default parameters
    uint32_t Q = 5;
    uint32_t B = 32;
    uint32_t NUM_BITS = 4;
    bool useFile = false;
    
    handleArgs(argc, argv, N, Q, B, NUM_BITS, useFile);

    const uint32_t H = 1 << NUM_BITS; // Number of buckets
    initHwd();
    
    // use the first CUDA device:
    hipSetDevice(0);
    
    unsigned int numblocks = (N + (Q * B - 1)) / (Q * B);
    uint32_t mem_size = N * sizeof(uint32_t);
    uint32_t hist_mem_size = numblocks * H * sizeof(uint32_t);
    printf("N is: %d\n", N);
    printf("Pred. Q: %d\n", Q);
    printf("Pred. B: %d\n", B);
    printf("Pred. b: %d\n", NUM_BITS);
    printf("====\n");
    printf("Num blocks: ceil(N / Q*B) = %d\n", numblocks);
    printf("H (RADIX): 2 ** b = %d\n", H);
    printf("====\n");
    printf("Memory size: %d\n", mem_size);
    printf("Histogram size: %d\n", hist_mem_size);
    printf("====\n");

    // allocate host memory for both CPU and GPU
    uint32_t* h_in  = (uint32_t*) malloc(mem_size);
    uint32_t* h_out = (uint32_t*) malloc(mem_size);
    uint32_t* gpu_res = (uint32_t*) malloc(hist_mem_size); // This can be removed later
    uint32_t* h_in_ref = (uint32_t*) malloc(mem_size);
    uint32_t* h_out_ref = (uint32_t*) malloc(mem_size);
    
    
    // initialize the memory
    srand(time(NULL));
    h_in_ref[0] = 1;
    h_in[0] = 1;
    if (VERBOSE) {
        printf("Input:\n");
        binaryPrinter(h_in[0], NUM_BITS);
        printf(", ");
    }
    for(unsigned int i=1; i<N; ++i) {
        // Chaining 4 rands to get 32-bit integer.
        h_in[i] = (rand() & 0xFF)
                | ((rand() & 0xFF) << 8)
                | ((rand() & 0xFF) << 16)
                | ((rand() & 0xFF) << 24); 
        h_in_ref[i] = h_in[i];
        if (VERBOSE) {
            binaryPrinter(h_in[i], NUM_BITS);       
            printf(", ");
        }
    }

    // allocate device memory
    uint32_t* d_in;
    uint32_t* d_out;
    uint32_t* d_hist;
    uint32_t* d_hist_scan;
    uint32_t* d_tmp; //REMOVE ME later (only used for shifting)
    uint32_t* d_in_ref;
    uint32_t* d_out_ref;
    uint32_t* d_hist_T; // Transposed histogram
    hipMalloc((void**)&d_in,  mem_size);
    hipMalloc((void**)&d_out, mem_size);
    hipMalloc((void**)&d_hist, hist_mem_size);
    hipMalloc((void**)&d_hist_scan, hist_mem_size);
    hipMalloc((void**)&d_tmp, hist_mem_size);
    hipMalloc((void**)&d_in_ref,  mem_size);
    hipMalloc((void**)&d_out_ref, mem_size);
    hipMalloc((void**)&d_hist_T, hist_mem_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);
    hipMemset(d_out, 0, mem_size);
    hipMemset(d_hist, 0, hist_mem_size);
    hipMemset(d_hist_scan, 0, hist_mem_size);
    hipMemcpy(d_in_ref, h_in_ref, mem_size, hipMemcpyHostToDevice);
    hipMemset(d_out_ref, 0, mem_size);

    // running Cub radix sort for reference
    struct timeval t_start, t_end, t_diff;
    uint64_t elapsed_cub = 0.0;
    for (int i = 0; i < GPU_RUNS; i++) {
        cubRadixSort(d_in_ref, d_out_ref, N, t_start, t_end);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_cub += (t_diff.tv_sec*1e6+t_diff.tv_usec);
    }
    elapsed_cub /= GPU_RUNS;
    
    // a small number of dry runs
    // for(int r = 0; r < 1; r++) {
    //     dim3 block(B, 1, 1), grid(numblocks, 1, 1);
    //     histogramKer<<< grid, block>>>(d_in, d_hist, mask, Q, N);
    // }
    const int W = sizeof(int) * 8; 
    const int num_passes = (W + NUM_BITS - 1) / NUM_BITS;
    unsigned int mask;

    uint64_t elapsed_cuda = 0.0;
    // We need to process numblocks * H elements in total
    // We have B threads per block
    // Therefore (numblocks * H) / B blocks are needed
    // (Ceil division)
    int hist_grid = (numblocks * H + B - 1) / B;
    uint32_t shift;
    for (int i = 0; i < GPU_RUNS; i++) {
        hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);
        hipMemset(d_out, 0, mem_size);
        hipMemset(d_hist, 0, hist_mem_size);
        hipMemset(d_hist_scan, 0, hist_mem_size);
        mask = (1 << NUM_BITS) - 1; // 4 bits = 0xF for radix 16
        gettimeofday(&t_start, NULL);

        for (int r = 0; r < num_passes; r++) {
            //The cpu does the following:
            //Holds the outer loop over passes (for pass in [0..num_passes))
            // Calculates mask and shift for each bit group
            // Launches the three GPU kernels per pass (histogram → scan → scatter) 
            // Swaps input/output pointers between passes

            // We also need to shift the bits accordingly after masking
            shift = r * NUM_BITS;

            //Allocates global memory buffers on device:
            // d_in, d_out for the arrays being sorted
            // d_histograms (size = numBlocks × H)
            // d_prefixes (prefix sums of histograms)
            // Performs small global memory resets (e.g. hipMemset)
            // Does NOT touch shared or register memory (that’s only inside kernels)
            histogramKer<<<numblocks, B, H * sizeof(uint32_t)>>>(d_in, d_hist, mask, shift, Q, N, H, NUM_BITS);
            
            callTransposeKer<32>(d_hist, d_hist_T, numblocks, H);

            // d_tmp is used as a temporary buffer to make d_hist ready for simulated exclusive scan
            // Should be removed
            shiftKer<<<hist_grid, B>>>(d_hist_T, d_tmp, numblocks * H);
            hipDeviceSynchronize();
              
            scanIncAddI32(B, numblocks * H, d_tmp, d_hist_T);

            callTransposeKer<32>(d_hist_T, d_hist_scan, H, numblocks);
            
            scatterKer<<<numblocks, B, H * sizeof(uint32_t)>>>(d_in, d_hist_scan, d_out, Q, N, mask, shift, H, NUM_BITS);
            hipDeviceSynchronize();

            // swap input and output arrays
            uint32_t* temp = d_in;
            d_in = d_out;
            d_out = temp;
        }
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_cuda += (t_diff.tv_sec*1e6+t_diff.tv_usec);
    }
    elapsed_cuda /= GPU_RUNS;
    
        
    // check for errors
    gpuAssert( hipPeekAtLastError() );

    // copy result from device to host
    hipMemcpy(gpu_res, d_hist, hist_mem_size, hipMemcpyDeviceToHost);

    // element-wise compare of CPU and GPU execution
    if (VERBOSE) {
    printf("\n\n-- Original histogram (transposed) -- ");
    for (int b = 0; b < H; b++) {
            printf("\n");
            for (int i = 0; i < numblocks; i++)
                printf("%u ", gpu_res[b * numblocks + i]);
        }
    }

    hipMemcpy(gpu_res, d_hist_scan, hist_mem_size, hipMemcpyDeviceToHost);
    
    // element-wise compare of CPU and GPU execution
    if (VERBOSE) {
        printf("\n\n-- Scanned histogram -- ");
        for (int b = 0; b < numblocks; b++) {
            printf("\n");
            for (int i = 0; i < H; i++)
                printf("%u ", gpu_res[b * H + i]);
        }
    }

    hipMemcpy(h_out, d_in, mem_size, hipMemcpyDeviceToHost);

    // element-wise compare of CPU and GPU execution
    if (VERBOSE) {
        printf("\n\n-- Result -- ");
            for (int i = 0; i < N; i++) 
            printf("%d ", h_out[i]);
    }

    // Verify correctness against Cub result
    hipMemcpy(h_out_ref, d_out_ref, mem_size, hipMemcpyDeviceToHost);
    bool validated = true;
    for (int i = 0; i < N; i++) {
        if (h_out[i] != h_out_ref[i]) {
            validated = false;
            break;
        }
    }
    if (validated) {
        printf("VALIDATED: Result matches CUB result\n");
    } else {
        printf("DID NOT VALIDATE: Result dont match CUB result!\nEXITING!\n");
        return 4;
    }
    printf("====\n");
    printf("CUB Radix Sort Time: %lu microseconds\n", elapsed_cub);
    printf("CUDA Radix Sort Time: %lu microseconds\n", elapsed_cuda);
    printf("====\n");
    
    // clean-up memory
    free(h_in);
    free(h_out);
    free(gpu_res); 
    free(h_in_ref);
    free(h_out_ref);
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_hist);
    hipFree(d_hist_scan);
    hipFree(d_tmp);
    hipFree(d_in_ref);
    hipFree(d_out_ref);
    hipFree(d_hist_T);
}

void handleArgs(int argc, char** argv, uint32_t& N, uint32_t& Q, uint32_t& B, uint32_t& NUM_BITS, bool& useFile) {
    // Reading the number of elements 
    if (argc < 2) { 
        printf("Missing N (number of elements) Exiting!\n");
        exit(1);
    }
    
    N = (uint32_t)atoi(argv[1]);

    const uint32_t maxN = 500000000;
    if(N > maxN) {
        printf("N is too big; maximal value is %d. Exiting!\n", maxN);
        exit(2);
    }

    // Optional arguments
    if (argc >= 3) {
        Q = (uint32_t)atoi(argv[2]);
    }

    if (argc >= 4) {
        B = (uint32_t)atoi(argv[3]);
    }

    if (argc >= 5) {
        NUM_BITS = (uint32_t)atoi(argv[4]);
    }

    if (argc >= 6) {
        bool useFile = (bool)atoi(argv[5]);
    }

    if (argc >= 7) {
        printf("Too many arguments! Exiting!\n");
        exit(3);
    }
}


template<int T>
void callTransposeKer(uint32_t* inp_d, uint32_t* out_d, const uint32_t height, const uint32_t width) {
    // inp_d : [height][width]uint32_t
    // out_d : [width][height]uint32_t (the transpose of inp_d.)

    // 1. setup block and grid parameters
    int  dimy = (height+T-1) / T; 
    int  dimx = (width +T-1) / T;
    dim3 block(T, T, 1);
    dim3 grid (dimx, dimy, 1);

    //2. execute the kernel
    coalsTransposeKer<T> <<< grid, block >>>(inp_d, out_d, height, width);
}

// Modified From assignment 2:
void scanIncAddI32(const uint32_t B, const size_t N, uint32_t* d_in, uint32_t* d_out) {
    // B: Desired CUDA block size (<= 1024, multiple of 32)
    // N: Length of the input array
    // d_in: Device input of size: N * sizeof(uint32_t)
    // d_out: device result of size: N * sizeof(uint32_t)
    uint32_t* d_tmp;
    hipMalloc((void**)&d_tmp, MAX_BLOCK*sizeof(uint32_t));
    hipMemset(d_out, 0, N*sizeof(uint32_t));

    scanInc<Add<uint32_t>> ( B, N, d_out, d_in, d_tmp );

    hipFree(d_tmp);
}

// Taken from CUB library examples
// https://nvidia.github.io/cccl/cub/api/structcub_1_1DeviceRadixSort.html

void cubRadixSort(uint32_t* d_in, uint32_t* d_out, size_t N, timeval& t_start, timeval& t_end) {
    //temporary storage
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, N);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // sorting operation
    gettimeofday(&t_start, NULL);
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, N);
    hipDeviceSynchronize();
    gettimeofday(&t_end, NULL);

    hipFree(d_temp_storage);
}

// Binary printer for debugging:
void binaryPrinter(int val, unsigned int decimal_points) {
    for (int i = decimal_points-1; i >= 0; i--) {
        if (val & (1 << i)) {
            printf("1");
        }
        else {
            printf("0");
        }
    }
}
